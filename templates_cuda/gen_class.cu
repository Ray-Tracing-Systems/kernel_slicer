
#include "LiteMath.h"
#include <extended/lm_device_vector.h> // also from LiteMath

namespace {{MainClassName}}{{MainClassSuffix}}_DEV
{
  {% for LocalFunc in LocalFunctions %} 
  {{LocalFunc}}
  {% endfor %}
  {% for Vector in VectorMembers %}
  __device__ LiteMathExtended::device_vector<{{Vector.DataType}}> {{Vector.Name}};
  {% endfor %}
  {% for Field in UBO.UBOStructFields %}
  {% if Field.IsDummy %} 
  __device__ uint {{Field.Name}}; 
  {% else %}
  {% if not Field.IsContainerInfo %}
  __device__ {{Field.Type}} {{Field.Name}}{% if Field.IsArray %}[{{Field.ArraySize}}]{% endif %};
  {% endif %}
  {% endif %}
  {% endfor %}
  
  {% for Kernel in KernelList %}
  __device__ void {{Kernel.Name}}({%for Arg in Kernel.OriginalArgs %}{{Arg.Type}} {{Arg.Name}}{% if loop.index != Kernel.LastArgAll %}, {% endif %}{% endfor %})
  {
    const uint _threadID[3] = {
      blockIdx.x * blockDim.x + threadIdx.x,
      blockIdx.y * blockDim.y + threadIdx.y,
      blockIdx.z * blockDim.z + threadIdx.z
    };
    {% for TID in Kernel.ThreadIds %}
    {% if TID.Simple %}
    const {{TID.Type}} {{TID.Name}} = {{TID.Type}}(_threadID[{{ loop.index }}]); 
    {% else %}
    const {{TID.Type}} {{TID.Name}} = {{TID.Start}} + {{TID.Type}}(_threadID[{{ loop.index }}])*{{TID.Stride}}; 
    {% endif %}
    {% endfor %}
    bool runThisThread = true;
    {% if not Kernel.EnableBlockExpansion %}
    {% if Kernel.threadDim == 3 %}
    if({{Kernel.threadName1}} >= {{Kernel.threadSZName1}} + {{Kernel.CondLE1}} || {{Kernel.threadName2}} >= {{Kernel.threadSZName2}} + {{Kernel.CondLE2}} || {{Kernel.threadName3}} >= {{Kernel.threadSZName3}} + {{Kernel.CondLE3}})
      runThisThread = false;
    {% else if Kernel.threadDim == 2 %}
    if({{Kernel.threadName1}} >= {{Kernel.threadSZName1}} + {{Kernel.CondLE1}} || {{Kernel.threadName2}} >= {{Kernel.threadSZName2}} + {{Kernel.CondLE2}})
      runThisThread = false;
    {% else %}
    if({{Kernel.threadName1}} >= {{Kernel.threadSZName1}} + {{Kernel.CondLE1}})
      runThisThread = false;
    {% endif %}
    {% endif %} {# /* if not Kernel.EnableBlockExpansion */ #}
    if(runThisThread) 
    {
    {{Kernel.Source}}
    }
  }

  {% endfor %}
};

#include <memory>
#include <cstdint>
#include <cassert>
#include <chrono>
#include "{{MainInclude}}"
{% for Include in AdditionalIncludes %}
#include "{{Include}}"
{% endfor %}

//#include <thrust/device_vector.h> // if use real thrust
//using thrust::device_vector;      // if use real thrust
using LiteMathExtended::device_vector;

class {{MainClassName}}{{MainClassSuffix}} : public {{MainClassName}}
{
public:

  {% for ctorDecl in Constructors %}
  {% if ctorDecl.NumParams == 0 %}
  {{ctorDecl.ClassName}}{{MainClassSuffix}}()
  {
    {% if HasPrefixData %}
    if({{PrefixDataName}} == nullptr)
      {{PrefixDataName}} = std::make_shared<{{PrefixDataClass}}>();
    {% endif %}
  }
  {% else %}
  {{ctorDecl.ClassName}}{{MainClassSuffix}}({{ctorDecl.Params}}) : {{ctorDecl.ClassName}}({{ctorDecl.PrevCall}})
  {
    {% if HasPrefixData %}
    if({{PrefixDataName}} == nullptr)
      {{PrefixDataName}} = std::make_shared<{{PrefixDataClass}}>();
    {% endif %}
  }
  {% endif %}
  {% endfor %}
  
  virtual ~{{MainClassName}}{{MainClassSuffix}}()
  {
    {% for Vector in VectorMembers %}
    {{Vector.Name}}_dev.resize(0);
    {{Vector.Name}}_dev.shrink_to_fit(); 
    {% endfor %}
  }

  void CommitDeviceData() override;

  void CopyUBOToDevice(bool a_updateVectorSize = true);
  void CopyUBOFromDevice();
  void UpdateDeviceVectors();

  {% for Kernel in Kernels %}
  void {{Kernel.OriginalDecl}} override;
  {% endfor %}
  
  {% for MainFunc in MainFunctions %}
  {{MainFunc.ReturnType}} {{MainFunc.MainFuncDeclCmd}} override;
  {% endfor %}

protected:
  {% for Vector in VectorMembers %}
  device_vector<{{Vector.DataType}}> {{Vector.Name}}_dev;
  {% endfor %}
};

{% for ctorDecl in Constructors %}
{% if ctorDecl.NumParams == 0 %}
std::shared_ptr<{{MainClassName}}> Create{{ctorDecl.ClassName}}{{MainClassSuffix}}()
{
  auto pObj = std::make_shared<{{MainClassName}}{{MainClassSuffix}}>();
  return pObj;
}
{% else %}
std::shared_ptr<{{MainClassName}}> Create{{ctorDecl.ClassName}}{{MainClassSuffix}}({{ctorDecl.Params}})
{
  auto pObj = std::make_shared<{{MainClassName}}{{MainClassSuffix}}>({{ctorDecl.PrevCall}});
  return pObj;
}
{% endif %}
{% endfor %}

void {{MainClassName}}{{MainClassSuffix}}::CopyUBOToDevice(bool a_updateVectorSize)
{
  {% for Var in ClassVars %}
  {% if Var.IsArray %}
  {% if Var.HasPrefix %}
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), pUnderlyingImpl->{{Var.CleanName}}, sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), {{Var.Name}}, sizeof({{Var.Name}}));
  {% endif %}
  {% else %}
  {% if Var.HasPrefix %}
  m_uboData.{{Var.Name}} = pUnderlyingImpl->{{Var.CleanName}};
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), &pUnderlyingImpl->{{Var.CleanName}}, sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), &{{Var.Name}}, sizeof({{Var.Name}}));
  {% endif %}
  {% endif %}
  {% endfor %}
  if(a_updateVectorSize)
  {
    using size_type = LiteMathExtended::device_vector<int>::size_type;
    {% for Var in ClassVectorVars %}
    {
      const size_type currSize = {{Var.Name}}_dev.size();
      hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_size), &currSize, sizeof(size_type));
    }
    {% endfor %}
  }
}

void {{MainClassName}}{{MainClassSuffix}}::CopyUBOFromDevice()
{
  //hipMemcpyFromSymbol(&h_globalVar, HIP_SYMBOL(globalVar), sizeof(int));
  {% for Var in ClassVars %}
  {% if Var.IsArray %}
  {% if Var.HasPrefix %}
  hipMemcpyFromSymbol(pUnderlyingImpl->{{Var.CleanName}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyFromSymbol({{Var.Name}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof({{Var.Name}}));
  {% endif %}
  {% else %}
  {% if Var.HasPrefix %}
  m_uboData.{{Var.Name}} = pUnderlyingImpl->{{Var.CleanName}};
  hipMemcpyFromSymbol(&pUnderlyingImpl->{{Var.CleanName}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyFromSymbol(&{{Var.Name}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof({{Var.Name}}));
  {% endif %}
  {% endif %}
  {% endfor %}
  using size_type = LiteMathExtended::device_vector<int>::size_type;
  {% for Var in ClassVectorVars %}
  {
    size_type currSize = 0;
    hipMemcpyFromSymbol(&currSize, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_size), sizeof(size_type));
    {{Var.Name}}.resize(currSize);
  }
  {% endfor %}
}

void {{MainClassName}}{{MainClassSuffix}}::UpdateDeviceVectors()
{
  using size_type = LiteMathExtended::device_vector<int>::size_type;
  {% for Var in VectorMembers %}
  {
    const size_type currSize = {{Var.Name}}_dev.size();
    const size_type currCapa = {{Var.Name}}_dev.capacity();
    const void*     currPtr  = {{Var.Name}}_dev.data();
    hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_data),     &currPtr,  sizeof(void*));
    hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_size    ), &currSize, sizeof(size_type));
    hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_capacity), &currCapa, sizeof(size_type));
  }
  {% endfor %}
}

void {{MainClassName}}{{MainClassSuffix}}::CommitDeviceData()
{ 
  {% for Var in VectorMembers %}
  {{Var.Name}}_dev.assign({{Var.Name}}.begin(), {{Var.Name}}.end());
  {% endfor %}
  UpdateDeviceVectors();
  CopyUBOToDevice(false);
}

{% for Kernel in Kernels %}
void {{MainClassName}}{{MainClassSuffix}}::{{Kernel.OriginalDecl}}
{
  dim3 block({{Kernel.WGSizeX}}, {{Kernel.WGSizeY}}, {{Kernel.WGSizeZ}});
  dim3 grid(({{Kernel.tidX}} + block.x - 1) / block.x, ({{Kernel.tidY}} + block.y - 1) / block.y, ({{Kernel.tidZ}} + block.z - 1) / block.z);
  {{MainClassName}}{{MainClassSuffix}}_DEV::{{Kernel.OriginalName}}<<<grid, block>>>({%for Arg in Kernel.OriginalArgs %}{{Arg.Name}}{% if loop.index != Kernel.LastArgAll %}, {% endif %}{% endfor %});
}

{% endfor %}
{% for MainFunc in MainFunctions %}
{{MainFunc.ReturnType}} {{MainClassName}}{{MainClassSuffix}}::{{MainFunc.MainFuncDeclCmd}}
{
  {% for var in MainFunc.FullImpl.InputData %}
  {{var.DataType}}* {{var.Name}}Host = {{var.Name}};
  {% endfor %}
  {% for var in MainFunc.FullImpl.OutputData %}
  {{var.DataType}}* {{var.Name}}Host = {{var.Name}};
  {% endfor %}

  {% for var in MainFunc.FullImpl.InputData %}
  hipMalloc(&{{var.Name}}, {{var.DataSize}}*sizeof({{var.DataType}}));
  {% endfor %}
  {% for var in MainFunc.FullImpl.OutputData %}
  hipMalloc(&{{var.Name}}, {{var.DataSize}}*sizeof({{var.DataType}}));
  {% endfor %}
  {% for var in MainFunc.FullImpl.InputData %}
  hipMemcpy((void*){{var.Name}}, {{var.Name}}Host, {{var.DataSize}}*sizeof({{var.DataType}}), hipMemcpyHostToDevice);
  {% endfor %}

  CopyUBOToDevice(true);
  {{MainFunc.MainFuncTextCmd}}
  CopyUBOFromDevice();
  {% for var in MainFunc.FullImpl.OutputData %}
  hipMemcpy({{var.Name}}Host, {{var.Name}}, {{var.DataSize}}*sizeof({{var.DataType}}), hipMemcpyDeviceToHost);
  {% endfor %}
  {% for var in MainFunc.FullImpl.InputData %}
  hipFree((void*){{var.Name}});
  {% endfor %}
  {% for var in MainFunc.FullImpl.OutputData %}
  hipFree({{var.Name}});
  {% endfor %}
}

{% endfor %}