#include "hip/hip_runtime.h"

#include "LiteMath.h"
#include <extended/lm_device_vector.h> // also from LiteMath
#include "{{MainInclude}}"
#include <cfloat>

namespace {{MainClassName}}{{MainClassSuffix}}_DEV
{
  using _Bool = bool;
  {% for Decl in ClassDecls %}
  {% if Decl.InClass and Decl.IsType %}
  using {{Decl.Type}} = {{MainClassName}}::{{Decl.Type}}; // for passing this data type to kernels
  {% endif %}
  {% endfor %}

  {% for LocalFunc in LocalFunctions %} 
  __device__ {{LocalFunc}}

  {% endfor %}
  {% for Vector in VectorMembers %}
  __device__ LiteMathExtended::device_vector<{{Vector.DataType}}> {{Vector.Name}};
  {% endfor %}
  {% for Field in UBO.UBOStructFields %}
  {% if Field.IsDummy %} 
  __device__ uint {{Field.Name}}; 
  {% else %}
  {% if not Field.IsContainerInfo %}
  __device__ {{Field.Type}} {{Field.Name}}{% if Field.IsArray %}[{{Field.ArraySize}}]{% endif %};
  {% endif %}
  {% endif %}
  {% endfor %}
  
  {% for MembFunc in AllMemberFunctions %}
  __device__ {{MembFunc.Decl}};
  {% endfor %}
  {% for MembFunc in AllMemberFunctions %}

  __device__ {{MembFunc.Text}}
  {% endfor %}
  {% if UseSubGroups %}
  template<typename T>
  __device__ void WarpReduceSum(volatile T* sdata, int tid) 
  {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
  }

  template<typename T>
  __device__ void WarpReduceMin(volatile T* sdata, int tid) 
  {
    sdata[tid] = min(sdata[tid], sdata[tid + 32]);
    sdata[tid] = min(sdata[tid], sdata[tid + 16]);
    sdata[tid] = min(sdata[tid], sdata[tid + 8]);
    sdata[tid] = min(sdata[tid], sdata[tid + 4]);
    sdata[tid] = min(sdata[tid], sdata[tid + 2]);
    sdata[tid] = min(sdata[tid], sdata[tid + 1]);
  }

  template<typename T>
  __device__ void WarpReduceMax(volatile T* sdata, int tid) 
  {
    sdata[tid] = max(sdata[tid], sdata[tid + 32]);
    sdata[tid] = max(sdata[tid], sdata[tid + 16]);
    sdata[tid] = max(sdata[tid], sdata[tid + 8]);
    sdata[tid] = max(sdata[tid], sdata[tid + 4]);
    sdata[tid] = max(sdata[tid], sdata[tid + 2]);
    sdata[tid] = max(sdata[tid], sdata[tid + 1]);
  }
  
  {% endif %}
  __device__ float atomicMin(float* address, float val) 
  {
    int* addr_as_int = (int*)address;
    int old = *addr_as_int;
    int expected;
    do {
        expected = old;
        float current_val = __int_as_float(old);
        if (val >= current_val) 
          break;  // Если новое значение не меньше, выходим
        old = atomicCAS(addr_as_int, expected, __float_as_int(val));
    } while (expected != old);
    return __int_as_float(old);
  }

  __device__ float atomicMax(float* address, float val) 
  {
    int* addr_as_int = (int*)address;
    int old = *addr_as_int;
    int expected;
    do {
        expected = old;
        float current_val = __int_as_float(old);
        if (val <= current_val) 
          break;  // Если новое значение не больше, выходим
        old = atomicCAS(addr_as_int, expected, __float_as_int(val));
    } while (expected != old);
    return __int_as_float(old);
  }
 
  {% for Kernel in KernelList %}
  __global__ void {{Kernel.Name}}({%for Arg in Kernel.OriginalArgs %}{{Arg.Type}} {{Arg.Name}}{% if loop.index != Kernel.LastArgAll %}, {% endif %}{% endfor %})
  {
    {% if not Kernel.IsSingleThreaded %}
    const uint _threadID[3] = {
      blockIdx.x * blockDim.x + threadIdx.x,
      blockIdx.y * blockDim.y + threadIdx.y,
      blockIdx.z * blockDim.z + threadIdx.z
    };
    {% for TID in Kernel.ThreadIds %}
    {% if TID.Simple %}
    const {{TID.Type}} {{TID.Name}} = {{TID.Type}}(_threadID[{{ loop.index }}]); 
    {% else %}
    const {{TID.Type}} {{TID.Name}} = {{TID.Start}} + {{TID.Type}}(_threadID[{{ loop.index }}])*{{TID.Stride}}; 
    {% endif %}
    {% endfor %}
    bool runThisThread = true;
    {% if not Kernel.EnableBlockExpansion %}
    {% if Kernel.IsIndirect %}
    {% if Kernel.threadDim == 3 %}
    if({{Kernel.threadName1}} >= {{Kernel.IndirectSizeX}} + {{Kernel.CondLE1}} || {{Kernel.threadName2}} >= {{Kernel.IndirectSizeY}} + {{Kernel.CondLE2}} || {{Kernel.threadName3}} >= {{Kernel.IndirectSizeZ}} + {{Kernel.CondLE3}})
      runThisThread = false;
    {% else if Kernel.threadDim == 2 %}
    if({{Kernel.threadName1}} >= {{Kernel.IndirectSizeX}} + {{Kernel.CondLE1}} || {{Kernel.threadName2}} >= {{Kernel.IndirectSizeY}} + {{Kernel.CondLE2}})
      runThisThread = false;
    {% else %}
    if({{Kernel.threadName1}} >= {{Kernel.IndirectSizeX}} + {{Kernel.CondLE1}})
      runThisThread = false;
    {% endif %}
    {% else %}
    {% if Kernel.threadDim == 3 %}
    if({{Kernel.threadName1}} >= {{Kernel.threadSZName1}} + {{Kernel.CondLE1}} || {{Kernel.threadName2}} >= {{Kernel.threadSZName2}} + {{Kernel.CondLE2}} || {{Kernel.threadName3}} >= {{Kernel.threadSZName3}} + {{Kernel.CondLE3}})
      runThisThread = false;
    {% else if Kernel.threadDim == 2 %}
    if({{Kernel.threadName1}} >= {{Kernel.threadSZName1}} + {{Kernel.CondLE1}} || {{Kernel.threadName2}} >= {{Kernel.threadSZName2}} + {{Kernel.CondLE2}})
      runThisThread = false;
    {% else %}
    if({{Kernel.threadName1}} >= {{Kernel.threadSZName1}} + {{Kernel.CondLE1}})
      runThisThread = false;
    {% endif %}
    {% endif %} {# /* if Kernel.IsIndirect  */ #}
    {% if length(Kernel.SubjToRed) > 0 or length(Kernel.ArrsToRed) > 0 %}                        
    {% include "inc_red_init.cu" %}
    {% endif %} 
    {% endif %} {# /* if not Kernel.EnableBlockExpansion */ #}
    if(runThisThread) 
    {
    {% endif %} {# /* if not Kernel.IsSingleThreaded */ #}
    {{Kernel.Source}}
    {% if not Kernel.EnableBlockExpansion and not Kernel.IsSingleThreaded %}
    }
    {% endif %}
    {% if Kernel.HasEpilog %}
    // GENERATED EPILOG:
    //
    {% if length(Kernel.SubjToRed) > 0 or length(Kernel.ArrsToRed) > 0 %}                      
    {% include "inc_red_finish.cu" %}
    {% endif %}
    {% endif %} {# /* END of 'if Kernel.HasEpilog'  */ #}
  }

  {% if Kernel.IsIndirect and not Kernel.IsSingleThreaded %}
  __global__ void {{Kernel.Name}}_Indirect({%for Arg in Kernel.OriginalArgs %}{{Arg.Type}} {{Arg.Name}}{% if loop.index != Kernel.LastArgAll %}, {% endif %}{% endfor %})
  {
    dim3 blocksNum, blockSize;
    blocksNum.x = ({{Kernel.IndirectSizeX}} - {{Kernel.IndirectStartX}} + {{Kernel.WGSizeX}} - 1)/{{Kernel.WGSizeX}};
    {% if Kernel.threadDim == 2 %}
    blocksNum.y = ({{Kernel.IndirectSizeY}} - {{Kernel.IndirectStartY}} + {{Kernel.WGSizeY}} - 1)/{{Kernel.WGSizeY}};
    {% endif %}
    {% if Kernel.threadDim == 3 %}
    blocksNum.z = ({{Kernel.IndirectSizeZ}} - {{Kernel.IndirectStartZ}} + {{Kernel.WGSizeZ}} - 1)/{{Kernel.WGSizeZ}};
    {% endif %}
    blockSize.x = {{Kernel.WGSizeX}};
    blockSize.y = {{Kernel.WGSizeY}};
    blockSize.z = {{Kernel.WGSizeZ}};
    {{Kernel.Name}}<<<blocksNum, blockSize>>>({%for Arg in Kernel.OriginalArgs %}{{Arg.Name}}{% if loop.index != Kernel.LastArgAll %}, {% endif %}{% endfor %});
  }
  
  {% endif %}
  {% endfor %}
};

#include <memory>
#include <cstdint>
#include <cassert>
#include <chrono>
#include <vector>
#include <string>
#include "{{MainInclude}}"
{% for Include in AdditionalIncludes %}
#include "{{Include}}"
{% endfor %}

//#include <thrust/device_vector.h> // if use real thrust
//using thrust::device_vector;      // if use real thrust
using LiteMathExtended::device_vector;

class {{MainClassName}}{{MainClassSuffix}} : public {{MainClassName}}
{
public:

  {% for ctorDecl in Constructors %}
  {% if ctorDecl.NumParams == 0 %}
  {{ctorDecl.ClassName}}{{MainClassSuffix}}()
  {
    {% if HasPrefixData %}
    if({{PrefixDataName}} == nullptr)
      {{PrefixDataName}} = std::make_shared<{{PrefixDataClass}}>();
    {% endif %}
  }
  {% else %}
  {{ctorDecl.ClassName}}{{MainClassSuffix}}({{ctorDecl.Params}}) : {{ctorDecl.ClassName}}({{ctorDecl.PrevCall}})
  {
    {% if HasPrefixData %}
    if({{PrefixDataName}} == nullptr)
      {{PrefixDataName}} = std::make_shared<{{PrefixDataClass}}>();
    {% endif %}
  }
  {% endif %}
  {% endfor %}
  
  virtual ~{{MainClassName}}{{MainClassSuffix}}()
  {
    {% for Vector in VectorMembers %}
    {{Vector.Name}}_dev.resize(0);
    {{Vector.Name}}_dev.shrink_to_fit(); 
    {% endfor %}
  }

  void CommitDeviceData() override;
  {% if HasGetTimeFunc %}
  void GetExecutionTime(const char* a_funcName, float a_out[4]) override;
  {% endif %}
  void CopyUBOToDevice(bool a_updateVectorSize = true);
  void CopyUBOFromDevice(bool a_updateVectorSize = true);
  void UpdateDeviceVectors();

  {% for Kernel in Kernels %}
  void {{Kernel.OriginalDecl}} override;
  {% endfor %}
  
  {% for MainFunc in MainFunctions %}
  {{MainFunc.ReturnType}} {{MainFunc.Name}}({%for Arg in MainFunc.InOutVarsAll %}{%if Arg.IsConst %}const {%endif%}{{Arg.Type}} {{Arg.Name}}{% if loop.index != MainFunc.InOutVarsLast %}, {% endif %}{% endfor %}) override;
  virtual {{MainFunc.ReturnType}} {{MainFunc.Name}}GPU({%for Arg in MainFunc.InOutVarsAll %}{%if Arg.IsConst %}const {%endif%}{{Arg.Type}} {{Arg.Name}}{% if loop.index != MainFunc.InOutVarsLast %}, {% endif %}{% endfor %});
  {% endfor %}

protected:
  {% for Vector in VectorMembers %}
  device_vector<{{Vector.DataType}}> {{Vector.Name}}_dev;
  {% endfor %}
  {% for MainFunc in MainFunctions %}
  float m_exTime{{MainFunc.Name}}[4] = {0,0,0,0};
  {% endfor %}
};

class {{MainClassName}}{{MainClassSuffix}}DEV : public {{MainClassName}}{{MainClassSuffix}}
{
public:

  {% for ctorDecl in Constructors %}
  {% if ctorDecl.NumParams == 0 %}
  {{ctorDecl.ClassName}}{{MainClassSuffix}}DEV() {}
  {% else %}
  {{ctorDecl.ClassName}}{{MainClassSuffix}}DEV({{ctorDecl.Params}}) : {{ctorDecl.ClassName}}({{ctorDecl.PrevCall}}) {}
  {% endif %}
  {% endfor %}
  {% for MainFunc in MainFunctions %}
  
  {{MainFunc.ReturnType}} {{MainFunc.Name}}({%for Arg in MainFunc.InOutVarsAll %}{%if Arg.IsConst %}const {%endif%}{{Arg.Type}} {{Arg.Name}}{% if loop.index != MainFunc.InOutVarsLast %}, {% endif %}{% endfor %}) override {
    {% if MainFunc.IsVoid %}
    {{MainFunc.Name}}GPU({%for Arg in MainFunc.InOutVarsAll %}{{Arg.Name}}{% if loop.index != MainFunc.InOutVarsLast %}, {% endif %}{% endfor %});
    {% else %}
    return {{MainFunc.Name}}GPU({%for Arg in MainFunc.InOutVarsAll %}{{Arg.Name}}{% if loop.index != MainFunc.InOutVarsLast %}, {% endif %}{% endfor %});
    {% endif %}
  }
  {% endfor %}

protected:
};

{% for ctorDecl in Constructors %}
{% if ctorDecl.NumParams == 0 %}
std::shared_ptr<{{MainClassName}}> Create{{ctorDecl.ClassName}}{{MainClassSuffix}}()
{
  auto pObj = std::make_shared<{{MainClassName}}{{MainClassSuffix}}>();
  return pObj;
}
{% else %}
std::shared_ptr<{{MainClassName}}> Create{{ctorDecl.ClassName}}{{MainClassSuffix}}({{ctorDecl.Params}})
{
  auto pObj = std::make_shared<{{MainClassName}}{{MainClassSuffix}}>({{ctorDecl.PrevCall}});
  return pObj;
}
{% endif %}
{% endfor %}
{% for ctorDecl in Constructors %}
{% if ctorDecl.NumParams == 0 %}
std::shared_ptr<{{MainClassName}}> Create{{ctorDecl.ClassName}}{{MainClassSuffix}}_DEV()
{
  auto pObj = std::make_shared<{{MainClassName}}{{MainClassSuffix}}DEV>();
  return pObj;
}
{% else %}
std::shared_ptr<{{MainClassName}}> Create{{ctorDecl.ClassName}}{{MainClassSuffix}}_DEV({{ctorDecl.Params}})
{
  auto pObj = std::make_shared<{{MainClassName}}{{MainClassSuffix}}DEV>({{ctorDecl.PrevCall}});
  return pObj;
}
{% endif %}
{% endfor %}

void {{MainClassName}}{{MainClassSuffix}}::CopyUBOToDevice(bool a_updateVectorSize)
{
  {% for Var in ClassVars %}
  {% if Var.IsArray %}
  {% if Var.HasPrefix %}
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), pUnderlyingImpl->{{Var.CleanName}}, sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), {{Var.Name}}, sizeof({{Var.Name}}));
  {% endif %}
  {% else %}
  {% if Var.HasPrefix %}
  m_uboData.{{Var.Name}} = pUnderlyingImpl->{{Var.CleanName}};
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), &pUnderlyingImpl->{{Var.CleanName}}, sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), &{{Var.Name}}, sizeof({{Var.Name}}));
  {% endif %}
  {% endif %}
  {% endfor %}
  if(a_updateVectorSize)
  {
    {% for Var in VectorMembers %}
    hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), &{{Var.Name}}_dev, sizeof(LiteMathExtended::device_vector<{{Var.DataType}}>));
    {% endfor %}
  }
}

void {{MainClassName}}{{MainClassSuffix}}::CopyUBOFromDevice(bool a_updateVectorSize)
{
  {% for Var in ClassVars %}
  {% if Var.IsArray %}
  {% if Var.HasPrefix %}
  hipMemcpyFromSymbol(pUnderlyingImpl->{{Var.CleanName}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyFromSymbol({{Var.Name}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof({{Var.Name}}));
  {% endif %}
  {% else %}
  {% if Var.HasPrefix %}
  m_uboData.{{Var.Name}} = pUnderlyingImpl->{{Var.CleanName}};
  hipMemcpyFromSymbol(&pUnderlyingImpl->{{Var.CleanName}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyFromSymbol(&{{Var.Name}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof({{Var.Name}}));
  {% endif %}
  {% endif %}
  {% endfor %}
  if(a_updateVectorSize)
  {
    {% for Var in VectorMembers %}
    hipMemcpyFromSymbol(&{{Var.Name}}_dev, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof(LiteMathExtended::device_vector<{{Var.DataType}}>));
    if({{Var.Name}}.size() != {{Var.Name}}_dev.size())
      {{Var.Name}}.resize({{Var.Name}}_dev.size());
    {% endfor %}
  }
}

void {{MainClassName}}{{MainClassSuffix}}::UpdateDeviceVectors() 
{
  {% for Var in VectorMembers %}
  {{Var.Name}}_dev.reserve({{Var.Name}}.capacity());
  {{Var.Name}}_dev.assign({{Var.Name}}.begin(), {{Var.Name}}.end());
  {% endfor %}
}

void {{MainClassName}}{{MainClassSuffix}}::CommitDeviceData()
{
  UpdateDeviceVectors();
  CopyUBOToDevice(true);
}

{% for Kernel in Kernels %}
void {{MainClassName}}{{MainClassSuffix}}::{{Kernel.OriginalDecl}}
{
  {% if Kernel.HasLoopInit %}
  {{MainClassName}}{{MainClassSuffix}}_DEV::{{Kernel.OriginalName}}_Init<<<1,1>>>({%for Arg in Kernel.OriginalArgs %}{{Arg.Name}}{% if loop.index != Kernel.LastArgAll %}, {% endif %}{% endfor %});
  {% endif %}
  {% if Kernel.IsIndirect %}
  {{MainClassName}}{{MainClassSuffix}}_DEV::{{Kernel.OriginalName}}_Indirect<<<1, 1>>>({%for Arg in Kernel.OriginalArgs %}{{Arg.Name}}{% if loop.index != Kernel.LastArgAll %}, {% endif %}{% endfor %});
  hipDeviceSynchronize(); // do we need to wait here? 
  {% else %}
  dim3 block({{Kernel.WGSizeX}}, {{Kernel.WGSizeY}}, {{Kernel.WGSizeZ}});
  dim3 grid(({{Kernel.tidX}} + block.x - 1) / block.x, ({{Kernel.tidY}} + block.y - 1) / block.y, ({{Kernel.tidZ}} + block.z - 1) / block.z);
  {{MainClassName}}{{MainClassSuffix}}_DEV::{{Kernel.OriginalName}}<<<grid, block>>>({%for Arg in Kernel.OriginalArgs %}{{Arg.Name}}{% if loop.index != Kernel.LastArgAll %}, {% endif %}{% endfor %});
  {% endif %}
  {% if Kernel.HasLoopFinish %}
  {{MainClassName}}{{MainClassSuffix}}_DEV::{{Kernel.OriginalName}}_Finish<<<1,1>>>({%for Arg in Kernel.OriginalArgs %}{{Arg.Name}}{% if loop.index != Kernel.LastArgAll %}, {% endif %}{% endfor %});
  {% endif %}
}

{% endfor %}
{% for MainFunc in MainFunctions %}
{{MainFunc.ReturnType}} {{MainClassName}}{{MainClassSuffix}}::{{MainFunc.Name}}GPU({%for Arg in MainFunc.InOutVarsAll %}{%if Arg.IsConst %}const {%endif%}{{Arg.Type}} {{Arg.Name}}{% if loop.index != MainFunc.InOutVarsLast %}, {% endif %}{% endfor %})
{
  {{MainFunc.MainFuncTextCmd}}
}

{{MainFunc.ReturnType}} {{MainClassName}}{{MainClassSuffix}}::{{MainFunc.Name}}({%for Arg in MainFunc.InOutVarsAll %}{%if Arg.IsConst %}const {%endif%}{{Arg.Type}} {{Arg.Name}}{% if loop.index != MainFunc.InOutVarsLast %}, {% endif %}{% endfor %})
{
  {% for var in MainFunc.FullImpl.InputData %}
  {{var.DataType}}* {{var.Name}}Host = {{var.Name}};
  {% endfor %}
  {% for var in MainFunc.FullImpl.OutputData %}
  {{var.DataType}}* {{var.Name}}Host = {{var.Name}};
  {% endfor %}
  
  hipEvent_t _start, _stop;
  hipEventCreate(&_start);
  hipEventCreate(&_stop);
  
  hipEventRecord(_start);
  {% for var in MainFunc.FullImpl.InputData %}
  hipMalloc(&{{var.Name}}, {{var.DataSize}}*sizeof({{var.DataType}}));
  {% endfor %}
  {% for var in MainFunc.FullImpl.OutputData %}
  hipMalloc(&{{var.Name}}, {{var.DataSize}}*sizeof({{var.DataType}}));
  {% endfor %}
  hipEventRecord(_stop);
  hipEventSynchronize(_stop);
  hipEventElapsedTime(&m_exTime{{MainFunc.Name}}[3], _start, _stop);
  
  hipEventRecord(_start);
  {% for var in MainFunc.FullImpl.InputData %}
  hipMemcpy((void*){{var.Name}}, {{var.Name}}Host, {{var.DataSize}}*sizeof({{var.DataType}}), hipMemcpyHostToDevice);
  {% endfor %}
  CopyUBOToDevice(true);
  hipEventRecord(_stop);
  hipEventSynchronize(_stop);
  hipEventElapsedTime(&m_exTime{{MainFunc.Name}}[1], _start, _stop);
  
  hipEventRecord(_start);
  {% if MainFunc.IsVoid %}
  {{MainFunc.Name}}GPU({%for Arg in MainFunc.InOutVarsAll %}{{Arg.Name}}{% if loop.index != MainFunc.InOutVarsLast %}, {% endif %}{% endfor %});
  {% else %}
  auto _resFromGPU = {{MainFunc.Name}}GPU({%for Arg in MainFunc.InOutVarsAll %}{{Arg.Name}}{% if loop.index != MainFunc.InOutVarsLast %}, {% endif %}{% endfor %});
  {% endif %}
  hipEventRecord(_stop);
  hipEventSynchronize(_stop);
  hipEventElapsedTime(&m_exTime{{MainFunc.Name}}[0], _start, _stop);
  
  hipEventRecord(_start);
  CopyUBOFromDevice();
  {% for var in MainFunc.FullImpl.OutputData %}
  hipMemcpy({{var.Name}}Host, {{var.Name}}, {{var.DataSize}}*sizeof({{var.DataType}}), hipMemcpyDeviceToHost);
  {% endfor %}
  hipEventRecord(_stop);
  hipEventSynchronize(_stop);
  hipEventElapsedTime(&m_exTime{{MainFunc.Name}}[2], _start, _stop);
  
  hipEventRecord(_start);
  {% for var in MainFunc.FullImpl.InputData %}
  hipFree((void*){{var.Name}});
  {% endfor %}
  {% for var in MainFunc.FullImpl.OutputData %}
  hipFree({{var.Name}});
  {% endfor %}
  hipEventRecord(_stop);
  hipEventSynchronize(_stop);
  float _timeForFree = 0.0f;
  hipEventElapsedTime(&_timeForFree, _start, _stop);
  m_exTime{{MainFunc.Name}}[3] += _timeForFree;
  hipEventDestroy(_start);
  hipEventDestroy(_stop);
  {% if not MainFunc.IsVoid %}
  return _resFromGPU;
  {% endif %}
}

{% endfor %}
{% if HasGetTimeFunc %}

void {{MainClassName}}{{MainClassSuffix}}::GetExecutionTime(const char* a_funcName, float a_out[4])
{
  {% for MainFunc in MainFunctions %}
  {% if MainFunc.OverrideMe %}
  if(std::string(a_funcName) == "{{MainFunc.Name}}" || std::string(a_funcName) == "{{MainFunc.Name}}Block")
  {
    a_out[0] = m_exTime{{MainFunc.Name}}[0];
    a_out[1] = m_exTime{{MainFunc.Name}}[1];
    a_out[2] = m_exTime{{MainFunc.Name}}[2];
    a_out[3] = m_exTime{{MainFunc.Name}}[3];
  }
  {% endif %}
  {% endfor %}
}
{% endif %}