
#include "LiteMath.h"
#include <extended/lm_device_vector.h> // also from LiteMath

namespace {{MainClassName}}{{MainClassSuffix}}_DEV
{
  {% for Vector in VectorMembers %}
  __device__ LiteMathExtended::device_vector<{{Vector.DataType}}> {{Vector.Name}};
  {% endfor %}
  {% for Field in UBO.UBOStructFields %}
  {% if Field.IsDummy %} 
  __device__ uint {{Field.Name}}; 
  {% else %}
  {% if not Field.IsContainerInfo %}
  __device__ {{Field.Type}} {{Field.Name}}{% if Field.IsArray %}[{{Field.ArraySize}}]{% endif %};
  {% endif %}
  {% endif %}
  {% endfor %}
  // if Pure CUDA, put kernels directly here
  {% for Kernel in Kernels %}
  //define {{Kernel.Name}} here ... 
  {% endfor %}
};

#include <memory>
#include <cstdint>
#include <cassert>
#include <chrono>
#include "{{MainInclude}}"
{% for Include in AdditionalIncludes %}
#include "{{Include}}"
{% endfor %}

//#include <thrust/device_vector.h> // if use real thrust
//using thrust::device_vector;      // if use real thrust
using LiteMathExtended::device_vector;

class {{MainClassName}}{{MainClassSuffix}} : public {{MainClassName}}
{
public:

  {% for ctorDecl in Constructors %}
  {% if ctorDecl.NumParams == 0 %}
  {{ctorDecl.ClassName}}{{MainClassSuffix}}()
  {
    {% if HasPrefixData %}
    if({{PrefixDataName}} == nullptr)
      {{PrefixDataName}} = std::make_shared<{{PrefixDataClass}}>();
    {% endif %}
  }
  {% else %}
  {{ctorDecl.ClassName}}{{MainClassSuffix}}({{ctorDecl.Params}}) : {{ctorDecl.ClassName}}({{ctorDecl.PrevCall}})
  {
    {% if HasPrefixData %}
    if({{PrefixDataName}} == nullptr)
      {{PrefixDataName}} = std::make_shared<{{PrefixDataClass}}>();
    {% endif %}
  }
  {% endif %}
  {% endfor %}
  
  void CommitDeviceData() override;

  void CopyUBOToDevice(bool a_updateVectorSize = true);
  void CopyUBOFromDevice();
  void UpdateDeviceVectors();

  {% for Kernel in Kernels %}
  void {{Kernel.OriginalDecl}} override;
  {% endfor %}
  
  {% for MainFunc in MainFunctions %}
  {{MainFunc.ReturnType}} {{MainFunc.MainFuncDeclCmd}} override;
  {% endfor %}

protected:
  {% for Vector in VectorMembers %}
  device_vector<{{Vector.DataType}}> {{Vector.Name}}_dev;
  {% endfor %}
};

{% for ctorDecl in Constructors %}
{% if ctorDecl.NumParams == 0 %}
std::shared_ptr<{{MainClassName}}> Create{{ctorDecl.ClassName}}{{MainClassSuffix}}()
{
  auto pObj = std::make_shared<{{MainClassName}}{{MainClassSuffix}}>();
  return pObj;
}
{% else %}
std::shared_ptr<{{MainClassName}}> Create{{ctorDecl.ClassName}}{{MainClassSuffix}}({{ctorDecl.Params}})
{
  auto pObj = std::make_shared<{{MainClassName}}{{MainClassSuffix}}>({{ctorDecl.PrevCall}});
  return pObj;
}
{% endif %}
{% endfor %}

void {{MainClassName}}{{MainClassSuffix}}::CopyUBOToDevice(bool a_updateVectorSize)
{
  {% for Var in ClassVars %}
  {% if Var.IsArray %}
  {% if Var.HasPrefix %}
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), pUnderlyingImpl->{{Var.CleanName}}, sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), {{Var.Name}}, sizeof({{Var.Name}}));
  {% endif %}
  {% else %}
  {% if Var.HasPrefix %}
  m_uboData.{{Var.Name}} = pUnderlyingImpl->{{Var.CleanName}};
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), &pUnderlyingImpl->{{Var.CleanName}}, sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), &{{Var.Name}}, sizeof({{Var.Name}}));
  {% endif %}
  {% endif %}
  {% endfor %}
  if(a_updateVectorSize)
  {
    using size_type = LiteMathExtended::device_vector<int>::size_type;
    {% for Var in ClassVectorVars %}
    {
      const size_type currSize = {{Var.Name}}_dev.size();
      hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_size), &currSize, sizeof(size_type));
    }
    {% endfor %}
  }
}

void {{MainClassName}}{{MainClassSuffix}}::CopyUBOFromDevice()
{
  //hipMemcpyFromSymbol(&h_globalVar, HIP_SYMBOL(globalVar), sizeof(int));
  {% for Var in ClassVars %}
  {% if Var.IsArray %}
  {% if Var.HasPrefix %}
  hipMemcpyFromSymbol(pUnderlyingImpl->{{Var.CleanName}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyFromSymbol({{Var.Name}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof({{Var.Name}}));
  {% endif %}
  {% else %}
  {% if Var.HasPrefix %}
  m_uboData.{{Var.Name}} = pUnderlyingImpl->{{Var.CleanName}};
  hipMemcpyFromSymbol(&pUnderlyingImpl->{{Var.CleanName}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof(pUnderlyingImpl->{{Var.CleanName}}));
  {% else %}
  hipMemcpyFromSymbol(&{{Var.Name}}, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}), sizeof({{Var.Name}}));
  {% endif %}
  {% endif %}
  {% endfor %}
  using size_type = LiteMathExtended::device_vector<int>::size_type;
  {% for Var in ClassVectorVars %}
  {
    size_type currSize = 0;
    hipMemcpyFromSymbol(&currSize, HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_size), sizeof(size_type));
    {{Var.Name}}.resize(currSize);
  }
  {% endfor %}
}

void {{MainClassName}}{{MainClassSuffix}}::UpdateDeviceVectors()
{
  using size_type = LiteMathExtended::device_vector<int>::size_type;
  {% for Var in VectorMembers %}
  {
    const size_type currSize = {{Var.Name}}_dev.size();
    const size_type currCapa = {{Var.Name}}_dev.capacity();
    const void*     currPtr  = {{Var.Name}}_dev.data();
    hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_data),     &currPtr,  sizeof(void*));
    hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_size    ), &currSize, sizeof(size_type));
    hipMemcpyToSymbol(HIP_SYMBOL({{MainClassName}}{{MainClassSuffix}}_DEV::{{Var.Name}}.m_capacity), &currCapa, sizeof(size_type));
  }
  {% endfor %}
}

void {{MainClassName}}{{MainClassSuffix}}::CommitDeviceData()
{ 
  {% for Var in VectorMembers %}
  {{Var.Name}}_dev.assign({{Var.Name}}.begin(), {{Var.Name}}.end());
  {% endfor %}
  UpdateDeviceVectors();
  CopyUBOToDevice(false);
}

{% for Kernel in Kernels %}
void {{MainClassName}}{{MainClassSuffix}}::{{Kernel.OriginalDecl}}
{
  // call actual kernel here
}

{% endfor %}
{% for MainFunc in MainFunctions %}
{{MainFunc.ReturnType}} {{MainClassName}}{{MainClassSuffix}}::{{MainFunc.MainFuncDeclCmd}}
{
  {% for var in MainFunc.FullImpl.InputData %}
  {{var.DataType}}* {{var.Name}}Host = {{var.Name}};
  {% endfor %}
  {% for var in MainFunc.FullImpl.OutputData %}
  {{var.DataType}}* {{var.Name}}Host = {{var.Name}};
  {% endfor %}

  {% for var in MainFunc.FullImpl.InputData %}
  hipMalloc(&{{var.Name}}, {{var.DataSize}}*sizeof({{var.DataType}}));
  {% endfor %}
  {% for var in MainFunc.FullImpl.OutputData %}
  hipMalloc(&{{var.Name}}, {{var.DataSize}}*sizeof({{var.DataType}}));
  {% endfor %}
  {% for var in MainFunc.FullImpl.InputData %}
  hipMemcpy((void*){{var.Name}}, {{var.Name}}Host, {{var.DataSize}}*sizeof({{var.DataType}}), hipMemcpyHostToDevice);
  {% endfor %}

  CopyUBOToDevice(true);
  {{MainFunc.MainFuncTextCmd}}
  CopyUBOFromDevice();
  {% for var in MainFunc.FullImpl.OutputData %}
  hipMemcpy({{var.Name}}Host, {{var.Name}}, {{var.DataSize}}*sizeof({{var.DataType}}), hipMemcpyDeviceToHost);
  {% endfor %}
}

{% endfor %}